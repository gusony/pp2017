/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define sqtau (1.0 * 0.3 / 1.0)*(1.0 * 0.3 / 1.0)


void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                   /* number of time steps */
    tpoints,                  /* total points along string */
    rcode;                    /* generic return code */
float  values[MAXPOINTS+2],   /* values at time t */
       oldval[MAXPOINTS+2],   /* values at time (t-dt) */
       newval[MAXPOINTS+2];   /* values at time (t+dt) */


/**********************************************************************
 * Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n",
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int j;
   float x, fac, k, tmp;
   time_t start = clock();

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0;
   tmp = tpoints - 1;

   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      oldval[j] = values[j];
      k = k + 1.0;
   }
   printf("init_line:%ld\n",clock() - start);
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   //float dtime, c, dx, tau, sqtau;

   /*dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;*/


   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}
__global__ void g_do_math(float *values, float *oldval, float *newval,int nsteps, int tpoints){
	int i = 0;
	int j = blockIdx.x*32 + threadIdx.x;

	for (i = 1; i<= nsteps; i++) {
		*(newval+1) = *(newval+tpoints) = 0.0;
		*(newval+j) = (2.0 * (*(values+j)) ) - (*(oldval+j)) + (sqtau *  (-2.0)* (*(values+j)) );

		*(oldval+j) = *(values+j);
		*(values+j) = *(newval+j);
	}
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
   	int blocknum = ((int)(tpoints/32))+1;

    float *d_values, *d_oldval, *d_newval;
    hipMalloc((void**)&d_values, sizeof(float) * (MAXPOINTS+2)); // values at time t
    hipMalloc((void**)&d_oldval, sizeof(float) * (MAXPOINTS+2)); // values at time (t-dt)
    hipMalloc((void**)&d_newval, sizeof(float) * (MAXPOINTS+2)); // values at time (t+dt)

    hipMemcpy(d_values, &values[0], sizeof(float) * MAXPOINTS+2, hipMemcpyHostToDevice);
	hipMemcpy(d_oldval, &oldval[0], sizeof(float) * MAXPOINTS+2, hipMemcpyHostToDevice);
	hipMemcpy(d_newval, &newval[0], sizeof(float) * MAXPOINTS+2, hipMemcpyHostToDevice);

	g_do_math<<<blocknum, 32>>>(d_values, d_oldval, d_newval, nsteps, tpoints);

	hipMemcpy(&values[0], d_values, sizeof(float) * MAXPOINTS+2, hipMemcpyDeviceToHost);
    hipMemcpy(&oldval[0], d_oldval, sizeof(float) * MAXPOINTS+2, hipMemcpyDeviceToHost);
	hipMemcpy(&newval[0], d_newval, sizeof(float) * MAXPOINTS+2, hipMemcpyDeviceToHost);
	hipFree(d_values);
	hipFree(d_oldval);
	hipFree(d_newval);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 * Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
   sscanf(argv[1],"%d",&tpoints);
   sscanf(argv[2],"%d",&nsteps);
   check_param();
   printf("Initializing points on the line...\n");
   init_line();
   printf("Updating all points for all time steps...\n");
   update();
   printf("Printing final results...\n");
   //printfinal();
   printf("\nDone.\n\n");

   return 0;
}